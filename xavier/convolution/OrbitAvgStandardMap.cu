#include "hip/hip_runtime.h"
/*************************************************************************
POMATO: POincare MAp TOpology: Extraction and visualization of the
        topological structure of the circular restricted three-body
        problem for orbital mechanics applications.

Authors: Wayne Schlei and Xavier Tricoche

Copyright (c) 2013-2018, Purdue University

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
**************************************************************************/

/*
  * OrbitAvgStandardMap.cu
  *  - Performs orbit averaging texture method for Standard Map
  *
  * Author:  Wayne Schlei
  * Date: 4/24/2012
  * Mod:  10/13/2014 - Update to CUDA 6.5
  *
  * Note:  Assumes a 2D grid. To Use:  call cudaOrbitAvgStandardMap();
*/

#include <stdio.h>
#include <stdlib.h>

// Include statements
//#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <hip/driver_types.h>
#include <driver_functions.h>
//Obsolete CUDA Utilities - deprecated in CUDA 6.5
//#include <cutil_inline.h>
//#include <cutil_math.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>
#include <helper_double_math.h>

// Constants
#define PI 3.1415926535897932

// GPU Textures
texture<float, 2,hipReadModeElementType>  texWhiteNoise;
texture<float4, 2, hipReadModeElementType> texRGBANoise;

// GPU constant memory
//Bounding Box(bbox.x = min_x, bbox.y = min_y, bbox.z = max_x, bbox.w = max_y)
__constant__ float min_x;
__constant__ float min_y;
__constant__ float max_x;
__constant__ float max_y;
__constant__ float k; //Chaos parameter
__constant__ float stretch; //A color stetching factor

//Debug helper message
void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }
}

//Setting Device constants
extern "C"
void cudaSetStandardMapConstants(float4 bounds, float chaosParam, float stretchFactor) {
	float minx,miny,maxx,maxy;
	minx = bounds.x; miny = bounds.y;
	maxx = bounds.z; maxy = bounds.w;
	//Copy constants from __host__ to __device__
	checkCudaErrors ( hipMemcpyToSymbol(HIP_SYMBOL(min_x),&minx,sizeof(float)) );
	checkCudaErrors ( hipMemcpyToSymbol(HIP_SYMBOL(min_y),&miny,sizeof(float)) );
	checkCudaErrors ( hipMemcpyToSymbol(HIP_SYMBOL(max_x),&maxx,sizeof(float)) );
	checkCudaErrors ( hipMemcpyToSymbol(HIP_SYMBOL(max_y),&maxy,sizeof(float)) );
	checkCudaErrors ( hipMemcpyToSymbol(HIP_SYMBOL(k),&chaosParam,sizeof(float)) );
	checkCudaErrors ( hipMemcpyToSymbol(HIP_SYMBOL(stretch),&stretchFactor,sizeof(float) ));
}

//Identify the position for a given pixel -> maybe not necessary
__device__ float2 getPosition( int2 pixel, const uint2 GridDims) {
	//Evaluate the spacing
	float2 h = make_float2( 0.0, 0.0);
	h.x = (max_x - min_x) / (float) (GridDims.x - 1);
	h.y = (max_y - min_y) / (float) (GridDims.y - 1);

	//Set Position as center of specified cell
	float2 pos = make_float2(
		min_x + (float) pixel.x * h.x + h.x/2.0,
		min_y + (float) pixel.y * h.y + h.y/2.0
	);

	return (pos);
}

//Identify the new pixel location
__device__ int2 getPixel(float2 pos, const uint2 GridDims) {
	//Evaluate the span of the grid
	float2 span = make_float2(max_x-min_x,max_y-min_y);

	//Move position inside of grid (modulo)
	//-> Maybe texture does it for me with wrap mode


	int2 pixel = make_int2(0,0);

	pixel.x = (int) ((pos.x/span.x)*(float)GridDims.x);
	pixel.y = (int) ((pos.y/span.y)*(float)GridDims.y);
	return (pixel);
}

//Running the Map in Forward Time
__device__ float2 advectStandardMapFwd( float2 pos ) {
	float2 newPos = make_float2( 0.0 , 0.0 );
	//EOMS -> But just Difference Equation
	newPos.y = pos.y + k / ( 2.0*PI ) * sin ( 2.0*PI * pos.x);
	newPos.x = pos.x + newPos.y;
	return (newPos);
}

//Modulous distance -  for Repeating maps like the Standard Map
__device__ float2 modulo(float2 pos) {
	//Transform
	float2 val = pos-make_float2(min_x,min_y);
	float2 span = make_float2(max_x-min_x,max_y-min_y);
	float2 r = make_float2(fmod(val.x,span.x),fmod(val.y,span.y)); //Remainder of val/span
	//Check x
	if (r.x==0.0) {
	 	val.x = pos.x - min_x;
	} else {
		val.x = (val.x>=0 ? r.x : span.x + r.x);
	}
	//Check y
	if (r.y==0.0) {
		val.y = pos.y - min_y;
	} else {
		val.y = (val.y>=0 ? r.y : span.y + r.y);
	}
	//Add min back on - Transform back
	val.x += min_x;
	val.y += min_y;
	return val;
}


__global__ void testkernel(float *imageOut, const int numIters, const uint2 GridDims) {
	unsigned int x = threadIdx.x + __umul24(blockIdx.x, blockDim.x);
	unsigned int y = threadIdx.y + __umul24(blockIdx.y, blockDim.y);

	if (x >= GridDims.x || y >= GridDims.y) return;
	int index = x+y*GridDims.x;
	imageOut[index] = ((float) index) * 0.001;
}

//Oribt-Avg Kernel: Gray channel
__global__ void orbit_average_stdMap_kernel(float *imageOutDevice, const int numIters, const uint2 GridDims) {
	unsigned int x = threadIdx.x + __umul24(blockIdx.x, blockDim.x);
	unsigned int y = threadIdx.y + __umul24(blockIdx.y, blockDim.y);

	if (x >= GridDims.x || y >= GridDims.y) return;

	//Get start position (normalized)
	int2 pixel = make_int2(x,y);
	float2 pos = getPosition(pixel, GridDims);

	//Transform Coords - Normalized
	float u = x / (float) GridDims.x;
	float v = y / (float) GridDims.y;


	//Loop for all iterations
	float acc = tex2D(texWhiteNoise,u,v);
	for (int ii=0; ii<numIters; ii++) {
		//Advect Standard Map
		float2 newPos = advectStandardMapFwd( pos );
		//Run Modulous math on position
		newPos = modulo(newPos);
		//Evaluate new pixel location in grid
		pixel = getPixel(newPos,GridDims);
		//Convert (int2) pixel to normalized coords
		u = pixel.x / (float) GridDims.x;
		v = pixel.y / (float) GridDims.y;
		//Get pixel color from white noise
		acc += tex2D(texWhiteNoise,u,v);
		pos = newPos;
	}

	//Evaluate the resulting color
	acc /= (float) (numIters+1);
	acc = (acc-0.5) * stretch + 0.5; //Apply stretch factor

	//Store resulting color
	int index = x+y*GridDims.x;
	imageOutDevice[index] = acc;

	//__syncthreads();
}

//Oribt-Avg Kernel: RGBA 4-channel
__global__ void orbit_average_stdMap_color_kernel(
	float *imageOutDevice, const int numIters, const uint2 GridDims) {
	unsigned int x = threadIdx.x + __umul24(blockIdx.x, blockDim.x);
	unsigned int y = threadIdx.y + __umul24(blockIdx.y, blockDim.y);

	if (x >= GridDims.x || y >= GridDims.y) return;

	//Get start position (normalized)
	int2 pixel = make_int2(x,y);
	float2 pos = getPosition(pixel, GridDims);

	//Transform Coords - Normalized
	float u = x / (float) GridDims.x;
	float v = y / (float) GridDims.y;


	//Loop for all iterations
	float4 acc = tex2D(texRGBANoise,u,v);
	for (int ii=0; ii<numIters; ii++) {
		//Advect Standard Map
		float2 newPos = advectStandardMapFwd( pos );
		//Run Modulous math on position
		newPos = modulo(newPos);
		//Evaluate new pixel location in grid
		pixel = getPixel(newPos,GridDims);
		//Convert (int2) pixel to normalized coords
		u = pixel.x / (float) GridDims.x;
		v = pixel.y / (float) GridDims.y;
		//Get pixel color from white noise
		acc += tex2D(texRGBANoise,u,v);
		pos = newPos;
	}

	//Evaluate the resulting color
	acc /= (float) (numIters+1);
	float4 half = make_float4(0.5f,0.5f,0.5f,0.5f);
	acc = (acc-half) * stretch + half; //Apply stretch factor

	//Store resulting color
	int index = x+y*GridDims.x;
	imageOutDevice[4*index+0] = acc.x;
	imageOutDevice[4*index+1] = acc.y;
	imageOutDevice[4*index+2] = acc.z;
	imageOutDevice[4*index+3] = acc.w;

	//__syncthreads();
}

//Format Descriptor for textures:
//Single value float:
//hipChannelFormatDesc channelDescFloat =
//		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
//float4:
//hipChannelFormatDesc channelDescFloat4 =
//		hipCreateChannelDesc(32,32,32,32,hipChannelFormatKindFloat);

//Texture-based Map Visualization for Standard Map:
//-------------------------------------------------------------------------------------
//Intensity only (black/white):
extern "C"
void cudaOrbitAvgStandardMap( float *imageOut, float *noiseData, const uint2 GridDims,const int numIters)
{
	//Block and thread dimensions
	dim3  threads( 16, 16, 1);
	dim3  blocks( (GridDims.x % threads.x !=0) ? (GridDims.x / threads.x + 1) : (GridDims.x / threads.x) ,
		      (GridDims.y % threads.y !=0) ? (GridDims.y / threads.y + 1) : (GridDims.y / threads.y) );

	//Allocate space for output data
	float *imageOutDevice;
	int imageSize = GridDims.x*GridDims.y;
	checkCudaErrors (  hipMalloc ( (void**)&imageOutDevice, imageSize * sizeof ( float ) ) );

	//Channel Descriptor
	hipChannelFormatDesc channelDescFloat =
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat); //Gray-scale only


	//Set Texture parameters ->Note:  These statements only work with cudaArrays or pitch linear memory (not hipMalloc)
	texWhiteNoise.addressMode[0] = hipAddressModeWrap; //Wraps texture so x=1.1 (out of grid) becomes x=0.1
	texWhiteNoise.addressMode[1] = hipAddressModeWrap;
	//texWhiteNoise.filterMode     = hipFilterModeLinear; //Linear interpolation between cells.
	texWhiteNoise.filterMode     = hipFilterModePoint; //Like Nearest-Neighbor
	texWhiteNoise.normalized     = true; //Normalized Coords (u,v on [0-1])


	//Host noise array
	//In this case, noiseData is already appropriately allocated
	//Allocate Noise array on Device
    hipArray *noiseDataDevice = 0;
    checkCudaErrors(hipMallocArray (&noiseDataDevice, &channelDescFloat, GridDims.x, GridDims.y));

	//Copy noiseData to device array
	checkCudaErrors (
	  hipMemcpyToArray(noiseDataDevice, 0, 0, noiseData, sizeof(float)*imageSize, hipMemcpyHostToDevice)
	);
	//Bind Array to Texture
	checkCudaErrors ( hipBindTextureToArray(texWhiteNoise, noiseDataDevice, channelDescFloat) );



	//Run Map and Orbit Averaging Procedure -> Returns a float4 per pixel for coloring
	orbit_average_stdMap_kernel<<<blocks,threads>>>(imageOutDevice,numIters,GridDims);
	checkCUDAError("run orbit_avg");
	//testkernel<<<blocks,threads>>>(imageOutDevice,numIters,GridDims);
	//checkCUDAError("run testkernel");

	//Copy Data from Device to Host
	checkCudaErrors ( hipMemcpy (imageOut, imageOutDevice, imageSize*sizeof(float), hipMemcpyDeviceToHost));

	//Unbind
	hipUnbindTexture( texWhiteNoise );
	//Free Device Memory
	checkCudaErrors ( hipFreeArray(noiseDataDevice) );
	checkCudaErrors ( hipFree((void*)imageOutDevice) );
}

//Color version -> 4D vector [r g b alpha]
extern "C"
void cudaColorOrbitAvgStandardMap( float *imageOut, float *noiseData, const uint2 GridDims,const int numIters)
{
	//Block and thread dimensions
	dim3  threads( 16, 16, 1);
	dim3  blocks( (GridDims.x % threads.x !=0) ? (GridDims.x / threads.x + 1) : (GridDims.x / threads.x) ,
		      (GridDims.y % threads.y !=0) ? (GridDims.y / threads.y + 1) : (GridDims.y / threads.y) );

	//Allocate space for output data
	float *imageOutDevice;
	int imageSize = GridDims.x*GridDims.y;
	checkCudaErrors (  hipMalloc ( (void**)&imageOutDevice, 4 * imageSize * sizeof ( float ) ) );

	//Channel Descriptor
	hipChannelFormatDesc channelDescFloat4 =
		hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat); //4D color+transparency

	//Set Texture parameters
	//Note:  These statements only work with cudaArrays or pitch linear memory (not hipMalloc)
	texRGBANoise.addressMode[0] = hipAddressModeWrap; //Wraps texture so x=1.1 (out of grid) becomes x=0.1
	texRGBANoise.addressMode[1] = hipAddressModeWrap;
	//texRGBANoise.filterMode     = hipFilterModeLinear; //Linear interpolation between cells.
	texRGBANoise.filterMode     = hipFilterModePoint; //Like Nearest-Neighbor
	texRGBANoise.normalized     = true; //Normalized Coords (u,v on [0-1])


	//Host noise array
	//In this case, noiseData is already appropriately allocated
	//Allocate Noise array on Device
    	hipArray *noiseDataDevice = 0;
    	checkCudaErrors(hipMallocArray (&noiseDataDevice, &channelDescFloat4, GridDims.x, GridDims.y));

	//Copy noiseData to device array
	checkCudaErrors (
	  hipMemcpyToArray(noiseDataDevice, 0, 0, noiseData, 4*imageSize*sizeof(float), hipMemcpyHostToDevice)
	);
	//Bind Array to Texture
	checkCudaErrors ( hipBindTextureToArray(texRGBANoise, noiseDataDevice, channelDescFloat4) );



	//Run Map and Orbit Averaging Procedure -> Returns a float4 per pixel for coloring
	orbit_average_stdMap_color_kernel<<<blocks,threads>>>(imageOutDevice,numIters,GridDims);
	checkCUDAError("run orbit_avg");
	//testkernel<<<blocks,threads>>>(imageOutDevice,numIters,GridDims);
	//checkCUDAError("run testkernel");

	//Copy Data from Device to Host
	checkCudaErrors ( hipMemcpy (imageOut, imageOutDevice, 4*imageSize*sizeof(float), hipMemcpyDeviceToHost));

	//Unbind
	hipUnbindTexture( texRGBANoise );
	//Free Device Memory
	checkCudaErrors ( hipFreeArray(noiseDataDevice) );
	checkCudaErrors ( hipFree((void*)imageOutDevice) );
}
